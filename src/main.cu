#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__
void calculateX(int *rowArr, int *colArr, double *valueArr,double *xArr,int n, int m)
{
  int dist = n/blockDim.x;
  if(n%blockDim.x > threadIdx.x)
    dist = dist + 1;

  int s = ((n%blockDim.x>(threadIdx.x-1))? dist : n/blockDim.x)*threadIdx.x;

  int i;
  double sum = 0;
  for(i = s; i< s + dist; i++)
    for(int j = rowArr[i]; j < ((i+1 < n)? rowArr[i+1] : m); j++)
      sum = sum + valueArr[j] * xArr[colArr[j]];
    xArr[i] = sum;
    sum = 0;
  }   

int main(int argc, char *argv[])
{
  int threads = atoi(argv[1]);
  int iterations = atoi(argv[2]);
  int answer = atoi(argv[3]);


  FILE *file = fopen(argv[4], "r");
  int num;
  double num2;
  int row;
  int col;

  fscanf(file, "%d", &num);
  int sizeOfMatrix = num;

  fscanf (file, "%d", &num);

  fscanf(file, "%d", &num);
  int numOfNumbers =  num;

  static double matrix[15000][15000];

  for(row = 0; row < sizeOfMatrix; row++)
    for(col = 0; col < sizeOfMatrix; col++)
      matrix[row][col] = 0;

  while(!feof (file))
  {
    fscanf(file, "%d", &num);
    row = num-1;
    fscanf(file, "%d", &num);
    col = num-1;
    fscanf(file, "%lf", &num2);

    matrix[row][col] = num2;
  }
  double *x = (double *)malloc(sizeOfMatrix*sizeof(double));
  
  for(row = 0; row < sizeOfMatrix; row++)
    x[row] = 1;

   int *row_ptr = (int *)malloc(sizeOfMatrix*sizeof(int));
   int *col_ind = (int *)malloc(numOfNumbers*sizeof(int));
   double *values = (double *)malloc(numOfNumbers*sizeof(double));

   int count = 0;
   int first = 0;

  for(row = 0; row < sizeOfMatrix; row++)
  {
    for(col = 0; col < sizeOfMatrix; col++)
    {
      if(matrix[row][col] != 0 && first == 0)
      {
        row_ptr[row] = count;
        col_ind[count] = col;
        values [count] = matrix[row][col];
        count ++;
        first = 1;
      }

      else if(matrix[row][col] != 0 && first == 1)
      {
        col_ind [count] = col;
        values [count] = matrix[row][col];
        count++;
      }

    }
    if(first == 0)
      row_ptr[row] = -1;
    
    first = 0;
  }

  int i = 1;
  for(row = 0; row < sizeOfMatrix; row++)
    if(row_ptr[row] == -1)
    {
      while(1)
      {
        if(row_ptr[row + i] != -1)
        {
          row_ptr[row] = row_ptr[row+i];
          break;
        }
        i++;
      }
      i = 1;
    }

  int *rowArr, *colArr;
  double *valueArr, *xArr;
  hipMalloc(&rowArr, sizeOfMatrix*sizeof(int));
  hipMalloc(&colArr, numOfNumbers*sizeof(int)); 
  hipMalloc(&valueArr, numOfNumbers*sizeof(double)); 
  hipMalloc(&xArr, sizeOfMatrix*sizeof(double));

  hipMemcpy(rowArr, row_ptr, sizeOfMatrix*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(colArr, col_ind, numOfNumbers*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(valueArr, values, numOfNumbers*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(xArr, x, sizeOfMatrix*sizeof(double), hipMemcpyHostToDevice);

  for(row = 0; row < iterations; row++)
  {
    calculateX<<<1, threads>>>(rowArr, colArr, valueArr, xArr, sizeOfMatrix, numOfNumbers);
    hipDeviceSynchronize();
  }

  hipMemcpy(x, xArr, sizeOfMatrix*sizeof(double), hipMemcpyDeviceToHost);


  if(answer == 1)
  {
    printf("ROW");
    for(row = 0; row < sizeOfMatrix; row++)
      printf("%d ",row_ptr[row]);

    printf("\n");
    printf("COL");
    printf("\n");
    for(row = 0; row < numOfNumbers; row++)
      printf("%d ",col_ind[row]);

    printf("\n");
    printf("VALUES");
    printf("\n");
    for(row = 0; row < numOfNumbers; row++)
      printf("%lf ",values[row]);

    printf("\n");
    printf("X ARRAY");
    printf("\n");
    for(row = 0; row < sizeOfMatrix; row++)
      printf("%lf ",x[row]);
  }

  hipFree(rowArr);
  hipFree(colArr);
  hipFree(valueArr);
  hipFree(xArr);

  free(row_ptr);
  free(col_ind);
  free(values);
  free(x);
}